#include "hip/hip_runtime.h"

#include "cmath"

#include <stdio.h>
#include <fstream>

int Height;
int Width;
int Seed;


hipError_t generatePerlineNoise(float* c);
bool SaveArrayInTxt(float* array);

__device__ float Frac(float xFloat)
{
    int xInt = fabs(xFloat);
    return fabs(xFloat) - xInt;
}

__device__ float Modulo(float a, float b) {
    int intaonb = a / b;
    return a - b * intaonb;
}

__device__ float Dot(float2 vectorLeft, float2 vectorRight)
{
    return vectorLeft.x * vectorRight.x + vectorLeft.y * vectorRight.y;
}

__device__ float Rand(float2 coordinate, int seed)
{
    float a = 12.9898;

    float b = 78.233;

    float c = 43758.5453;

    float dt = (coordinate.x + seed) * a + (coordinate.y + seed) * b;

    float sn = Modulo(dt, 2 * 3.141592653589793f);

    return Frac(sin(sn) * c);
}

__device__ float Rand(int2 xInt)
{
    float2 xFloat;
    xFloat.x = xInt.x;
    xFloat.y = xInt.y;
    return Rand(xFloat, 5);
}

__device__ float GenerateNoiseWithResolution(int2 uv)
{
    float randomNumber = Rand(uv);
    return randomNumber;
}

__device__ double cubicInterpolate(double p[4], double x) {
    return p[1] + 0.5 * x * (p[2] - p[0] + x * (2.0 * p[0] - 5.0 * p[1] + 4.0 * p[2] -
        p[3] + x * (3.0 * (p[1] - p[2]) + p[3] - p[0])));
}

__device__ double bicubicInterpolate(double p[4][4], double x, double y) {
    double arr[4];
    arr[0] = cubicInterpolate(p[0], y);
    arr[1] = cubicInterpolate(p[1], y);
    arr[2] = cubicInterpolate(p[2], y);
    arr[3] = cubicInterpolate(p[3], y);
    return cubicInterpolate(arr, x);
}

__device__ float BilinearInterpolation(float f00, float f01, float f10, float f11, int x0, int x1, int y0, int y1, float2 uv)
{
    float fR1 = (x1 - uv.x) / (x1 - x0) * f00 + (uv.x - x0) / (x1 - x0) * f10;
    float fR2 = (x1 - uv.x) / (x1 - x0) * f01 + (uv.x - x0) / (x1 - x0) * f11;
    return (y1 - uv.y) / (y1 - y0) * fR1 + (uv.y - y0) / (y1 - y0) * fR2;
}

__device__ float GenerateOctaveWithBicubic(float2 uv, int coeficient, int seed)
{
    double p[4][4];
    float u, v;

    for (int i = 0; i < 4; i++) {
        for (int j = 0; j < 4; j++) {
            float2 coordCoef = make_float2(0, 0);
            coordCoef.x = float(int(uv.x / (1.f / coeficient)) + i - 1) / coeficient + coeficient;
            coordCoef.y = float(int(uv.y / (1.f / coeficient)) + j - 1) / coeficient + coeficient;
            p[i][j] = Rand(coordCoef, seed);
        }
    }

    u = (uv.x - float(int(uv.x / (1.f / coeficient))) / coeficient) / (float(int(uv.x / (1.f / coeficient)) + 1) / coeficient - float(int(uv.x / (1.f / coeficient))) / coeficient);
    v = (uv.y - float(int(uv.y / (1.f / coeficient))) / coeficient) / (float(int(uv.y / (1.f / coeficient)) + 1) / coeficient - float(int(uv.y / (1.f / coeficient))) / coeficient);

    return bicubicInterpolate(p, u, v);
}

__device__ float GenerateOctaveWithBilinear(float2 uv, int coeficient)
{
    return BilinearInterpolation(GenerateNoiseWithResolution(make_int2(uv.x * coeficient, uv.y * coeficient)), GenerateNoiseWithResolution(make_int2(uv.x * coeficient, uv.y * coeficient + 1)),
        GenerateNoiseWithResolution(make_int2(uv.x * coeficient + 1, uv.y * coeficient)), GenerateNoiseWithResolution(make_int2(uv.x * coeficient + 1, uv.y * coeficient + 1)), uv.x * coeficient,
        uv.x * coeficient + 1, uv.y * coeficient, uv.y * coeficient + 1, make_float2(uv.x * coeficient, uv.y * coeficient));
}

__device__ float PerlinNoise(float2 uv, int seed)
{
    float color = 0.f;
    int numberOfCycles = 7;
    for (int i = numberOfCycles; i >= 1; i--)
    {
        color += GenerateOctaveWithBicubic(uv, pow(2, i), seed) / pow(2, i);
    }
    if (color > 1.f)
        color = 1.f;
    return color;
}

__device__ float Corrector(float2 uv)
{
    float corrector = 1.f - ((uv.x - 0.5f) * (uv.x - 0.5f) + (uv.y - 0.5f) * (uv.y - 0.5f));
    if (corrector < 0) corrector = 0.f;
    return corrector * corrector;
}

__global__ void addKernel(float* c, int* height, int* width, int* seed)
{
    int i = threadIdx.x;
    int j = blockIdx.x;
    float corrector = Corrector(make_float2((float)i / height[0], (float)j / width[0]));
    float perlinNoise = PerlinNoise(make_float2((float)i / height[0], (float)j / width[0]), seed[0]);
    c[i * width[0] + j] = pow(perlinNoise, 2);
}

int GenerateAndSavePerlineNoise(int height, int width, int seed)
{
    Height = height;
    Width = width;
    Seed = seed;

    float* c = new float[Height * Width];

    hipError_t cudaStatus = generatePerlineNoise(c);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Main function of Cuda failed!");
        return 1;
    }

    SaveArrayInTxt(c);

    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

hipError_t generatePerlineNoise(float* c)
{
    float* dev_c = 0;
    int height[2]; height[0] = Height; height[1] = Height;
    int width[2]; width[0] = Width; width[1] = Width;
    int seed[2]; seed[0] = Seed; seed[1] = Seed;
    int* dev_height;
    int* dev_width;
    int* dev_seed;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_c, Height * Width * sizeof(float));
    hipMalloc((void**)&dev_height, 2 * sizeof(int));
    hipMalloc((void**)&dev_width, 2 * sizeof(int));
    hipMalloc((void**)&dev_seed, 2 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    hipMemcpy(dev_height, height, 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_width, width, 2 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_seed, seed, 2 * sizeof(int), hipMemcpyHostToDevice);


    addKernel <<< Height, Width >>> (dev_c, dev_height, dev_width, dev_seed);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, Height * Width * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);

    return cudaStatus;
}

bool SaveArrayInTxt(float* arrayOfElements)
{
    FILE* file;
    if ((file = fopen("perlineNoise.txt", "w")) == NULL) {
        printf("error\n");
        return false;
    }
    else {
        for (int i = 0; i < Height; i++) {
            for (int j = 0; j < Width; j++)
                fprintf(file, "%f\t", arrayOfElements[i * Width + j]);
            fprintf(file, "\n");
        }
    }
    fclose(file);
    printf("Successful \n");
    return true;
}
